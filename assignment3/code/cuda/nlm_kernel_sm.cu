/**************************************************************
* Implementation of non local means using gpu,
* In this implementation shared memory is used
* to achieve the maximum speed up
**************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define PSIZE 3

//declaration of functions

//****host functions****
float **read_image(char *im_path,  int im_size);
void    write_image2(char *d_path , float  *im, int im_size);
float  *gaussian_filter(int patch_size, float sigma);
float  *to_rowmajor(float **image, int im_size);
float **add_padding(float **image, int im_size, int p_size, int *pad_size);
void   checkCuda(hipError_t result);

//****device functions****
__global__ void filter_kernel(float *image, float *denoised_image, float *gauss_kernel, int p_size, int im_size, float f_s );
__device__ void find_patch(float *pixel_patch, int current_pixel,int im_size,int p_size,float *image);
__device__ float compute_weight(float *pixel_patch,float *temp_patch,float *g_kernel, int p_size, float f_sigma);

int main(int argc, char *argv[]){

  //pass all parameters needed for computation.
  //patch size now is defined because must be known in compile time
  if(argc !=5){
    printf("Give correct arguments:1)source file(.csv format), 2) image size, 3)filter sigma, 4)gaussian filter sigma\n");
    exit(1);
  }

  char *image_path      = argv[1];
  int   image_size      = atoi(argv[2]);
  float f_sigma         = atof(argv[3]);
  float g_sigma         = atof(argv[4]);
  int   patch_size      = PSIZE;
  float *den_image      = (float *)malloc(image_size*image_size*sizeof(float));

  //read 2-D image
  float **I = read_image(image_path,image_size);

  //add padding
  int padded_size;
  float **D=add_padding(I,image_size,patch_size,&padded_size);

  //the initial 2-D image is now useless
  for(int i=0;i<image_size;i++)free(I[i]);
  free(I);

  //write image to rowmajor format
  float *padded_I = to_rowmajor(D,padded_size);

  //the padded 2-D image is now useless
  for(int i=0;i<padded_size;i++)free(D[i]);
  free(D);

  //compute gaussian filte
  float *GKERNEL = gaussian_filter(patch_size,g_sigma);

  //******** gpu section ********

  //define block and grid size
  int block_size = 256;
  int grid_size = image_size*image_size/block_size;

  //device variables
  float *dev_padded_image, *dev_denoised_image, *dev_g_kernel;

  //variables to compute time of execution.
  hipEvent_t start,stop;

  checkCuda(hipEventCreate(&start));
  checkCuda(hipEventCreate(&stop));

  checkCuda(hipMalloc((void **)&dev_g_kernel,patch_size*patch_size*sizeof(float)));
  checkCuda(hipMemcpy(dev_g_kernel,GKERNEL,patch_size*patch_size*sizeof(float),hipMemcpyHostToDevice));

  checkCuda(hipMalloc((void **)&dev_padded_image,padded_size*padded_size*sizeof(float)));
  checkCuda(hipMemcpy(dev_padded_image,padded_I,padded_size*padded_size*sizeof(float),hipMemcpyHostToDevice));

  checkCuda(hipMalloc((void **)&dev_denoised_image,image_size*image_size*sizeof(float)));

  /*************************nlm execution*************************/
  //declare the size for shared memory.
  int shared_mem_size = padded_size*(block_size/image_size + 2*(patch_size/2));
  hipEventRecord(start);
  filter_kernel<<<grid_size,block_size,shared_mem_size*sizeof(float)>>>(dev_padded_image,dev_denoised_image,dev_g_kernel,patch_size,padded_size,f_sigma);
  hipEventRecord(stop);
  /***************************************************************/

  checkCuda(hipMemcpy(den_image,dev_denoised_image,image_size*image_size*sizeof(float),hipMemcpyDeviceToHost));

  checkCuda(hipEventSynchronize(stop));
  float milliseconds = 0;
  checkCuda(hipEventElapsedTime(&milliseconds,start,stop));


  checkCuda(hipEventDestroy(start));
  checkCuda(hipEventDestroy(stop));
  checkCuda(hipFree(dev_g_kernel));
  checkCuda(hipFree(dev_padded_image));
  checkCuda(hipFree(dev_denoised_image));

  //some print messages
  printf("*************************************************************************************\n");
  printf("gpu execution (use of shared memory) finished. Total time is %f msec\n",milliseconds);
  printf("image size is %d\n", image_size);
  printf("patch size is %d\n", patch_size);
  printf("filtes sigma is %f and gauss kernel sigma is %f\n",f_sigma,g_sigma);
  printf("**************************************************************************************\n");

  char denoised_path[200];
  snprintf(denoised_path,sizeof(denoised_path),"../data/gpu_sm_denoised_image_%d_%d.csv",image_size,patch_size);
  write_image2(denoised_path,den_image,image_size);

  free(GKERNEL);
  free(den_image);
  free(padded_I);

}



/***********host functions****************/

float **add_padding(float **image, int im_size, int p_size, int *pad_size){

  int padding     = (p_size-1)/2;
  int padded_size = im_size + padding*2;
  *pad_size = padded_size;

  //allocate memomry for padded image
  float **padded_image =(float **)malloc(padded_size*sizeof(float *));
  for(int i=0;i<padded_size;i++)padded_image[i]=(float *)malloc(padded_size*sizeof(float));

  //initialize all values to -1 for debugging puproses...Later this will not be necessary
  for(int i=0;i<padded_size;i++){
    for(int j=0;j<padded_size;j++){
      padded_image[i][j]=-1;
    }
  }

  //first copy the body of the padded array which will remain the same
  //and also compute outter pixels through mirroring.
  for(int row = padding;row<padding+im_size;row++){
    for(int col=0;col<padded_size;col++){
      if(col<padding)padded_image[row][col] = image[row-padding][padding-col-1];
      else if(col>=padding+im_size)padded_image[row][col] = image[row-padding][im_size-1-(col-padding-im_size)];
      else padded_image[row][col] = image[row-padding][col-padding];
    }
  }

  //mirroring first and last rows.
  for(int row=0;row<padding;row++){
    for(int col = 0;col<padded_size;col++)padded_image[row][col] = padded_image[padding+(padding-row-1)][col];
  }
  for(int row=padding+im_size;row<padded_size;row++){
    for(int col=0;col<padded_size;col++)padded_image[row][col] = padded_image[padding+im_size-1-(row-padding-im_size)][col];
  }


  return padded_image;
}


float *gaussian_filter(int patch_size, float sigma){

  //allocate memory for the kernel
  float *kernel   = (float *)malloc(patch_size*patch_size*sizeof(float ));

  //compute variance
  float s         = 2.0*sigma*sigma;

  //sum for normalization
  float sum       = 0.0;

  int boundary    = patch_size/2;
  float r;

  for(int x=-boundary; x<=boundary; x++){
    for(int y=-boundary; y<=boundary; y++){
      r = x*x + y*y;
      kernel[(x+boundary)*patch_size+(y+boundary)] = exp(-r/s);
      sum += kernel[(x+boundary)*patch_size+(y+boundary)];
    }
  }

  //find the max and alos divide with the total sum
  float max = -1;
  for(int x=0;x<patch_size;x++){
    for(int y=0;y<patch_size;y++){
      kernel[x*patch_size+y]= kernel[x*patch_size+y]/sum;
      if(kernel[x*patch_size+y]>max) max = kernel[x*patch_size+y];
    }
  }

  //devide also with the max value
  for(int x=0;x<patch_size;x++){
    for(int y=0;y<patch_size;y++){
      kernel[x*patch_size+y]= kernel[x*patch_size+y]/max;
    }
  }


  return kernel;
}


float *to_rowmajor(float **image, int im_size){

  //allocate memory
  float *row_image = (float *)malloc(im_size*im_size*sizeof(float));

  //copy pixels to 1-D array.
  for(int row=0;row<im_size;row++){
    for(int col=0;col<im_size;col++){
      row_image[row*im_size+col] = image[row][col];
    }
  }

  return row_image;
}


void checkCuda(hipError_t result){
  if(result != hipSuccess){
    printf("CUDA Runtime Error :%s\n",hipGetErrorString(result));
    exit(1);
  }
}


float **read_image(char *im_path,int im_size){

  //open image file
  FILE *file = fopen(im_path,"r");

  //allocate memory
  float **image = (float **)malloc(im_size*sizeof(float *));
  for(int i=0;i<im_size;i++)image[i] = (float *)malloc(im_size*sizeof(float));

  //read the image
  for(int i=0;i<im_size;i++){
    for(int j=0;j<im_size;j++){
      fscanf(file,"%f,",image[i]+j);
    }
  }

  fclose(file);
  return image;
}


void   write_image2(char *d_path , float  *im, int im_size){

  //open file
  FILE *file = fopen(d_path,"w");

  //write image to file
  for(int i=0;i<im_size;i++){
    for(int j=0;j<im_size;j++){
      fprintf(file,"%f,",im[i*im_size +j]);
    }
    fprintf(file,"\n");
  }

  fclose(file);
}


/***********device functions****************/

__global__ void filter_kernel(float *image, float *denoised_image, float *gauss_kernel, int p_size, int im_size, float f_s ){

  //find the id for every thread
  int id = blockIdx.x*blockDim.x+threadIdx.x;

  //the shared memory must have all the pixels for every thread in the block
  //and also the padding in order to compute patches from here without the need
  //to look in global mem.
  extern __shared__ float image_section[];

  //save gauss kernel to shared memory, because we access it very often
  __shared__ float g_k[PSIZE*PSIZE];

  //for every thread calculate which pixel will compute. Taken into account the padding.
  int padded_pixel  = im_size*(p_size-1)/2+blockDim.x/(im_size-2*(p_size-1)/2)*blockIdx.x*im_size+
                      im_size*(threadIdx.x/(im_size-2*(p_size-1)/2))+threadIdx.x%(im_size-2*(p_size-1)/2)+(p_size-1)/2;


  //find for every pixel the corresponding patch
  float pixel_patch[PSIZE*PSIZE];
  find_patch(pixel_patch,padded_pixel,im_size,p_size,image);
  float  temp_patch[PSIZE*PSIZE];

  //write gauss kernel on shared mem.
  if(threadIdx.x == 0){
    for(int i=0;i<p_size;i++){
      for(int j=0;j<p_size;j++)
        g_k[i*p_size+j] = gauss_kernel[i*p_size+j];
    }
  }

  denoised_image[id] = -1;
  float Z =0;
  float W;
  float estimation=0;
  int temp_pixel;

  //load first elements on shared memory
  int shared_rows = blockDim.x/(im_size-2*(p_size-1)/2) + 2*(p_size-1)/2;
  int step = blockDim.x/(im_size-2*(p_size-1)/2);

  if(threadIdx.x<shared_rows){
    for(int i=0;i<im_size;i++)
      image_section[threadIdx.x*im_size+i]=image[threadIdx.x*im_size+i];
  }
  __syncthreads();

  for(int row=(p_size-1)/2;row<im_size-(p_size-1)/2;row = row+step){

    for(int s_row=(p_size-1)/2;s_row<shared_rows-(p_size-1)/2;s_row++){
      for(int s_col=(p_size-1)/1;s_col<im_size-(p_size-1)/2;s_col++){
        temp_pixel = s_row*im_size+s_col;
        find_patch(temp_patch,temp_pixel,im_size,p_size,image_section);
        W = compute_weight(pixel_patch,temp_patch,g_k,p_size,f_s);
        Z = Z + W;
        estimation = estimation + W*image_section[temp_pixel];
      }
    }

    //shift last elements
    __syncthreads();
    if(threadIdx.x == 0){
      for(int i=step;i<shared_rows;i++){
        for(int j=0;j<im_size;j++)
          image_section[im_size*(i-step)+j] = image_section[i*im_size+j];
      }
    }

    //bring the new elements in shared memory shifting the window steps time ahead
    __syncthreads();
    if(threadIdx.x>=shared_rows-step && threadIdx.x<shared_rows){
      for(int k=0;k<im_size;k++)
        image_section[threadIdx.x*im_size+k]=image[(row+step+threadIdx.x-(p_size-1)/2)*im_size+k];
    }
    __syncthreads();
  }

  denoised_image[id] = estimation/Z;
}


__device__ void find_patch(float *pixel_patch, int current_pixel,int im_size,int p_size,float *image){
  int boundary = (p_size-1)/2;
  for(int i=-boundary;i<=boundary;i++){
    for(int j=-boundary;j<=boundary;j++){
      pixel_patch[(i+boundary)*p_size+(j+boundary)] = image[current_pixel+i*im_size+j];
    }
  }
}


__device__ float compute_weight(float *pixel_patch,float *temp_patch,float *g_kernel, int p_size, float f_sigma){
  float temp=0;
  for(int i=0;i<p_size*p_size;i++) temp = temp + g_kernel[i]*g_kernel[i]*(pixel_patch[i]-temp_patch[i])*(pixel_patch[i]-temp_patch[i]);
  return exp(-temp/(f_sigma*f_sigma));
}
