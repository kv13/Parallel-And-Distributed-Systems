/**************************************************************
* Implementation of non local means using gpu,
* In this implementation shared memory isn't used.
* The algorithm follows the matlab code was given.
**************************************************************/



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define PSIZE 3


//declaration of functions

//****host functions****
void    write_image2(char *d_path , float  *im, int im_size);
float **read_image(char *im_path,  int im_size);
float **add_padding(float **image, int im_size, int p_size, int *pad_size);
float  *gaussian_filter(int patch_size, float sigma);
float  *to_rowmajor(float **image, int im_size);
void    checkCuda(hipError_t result);

//****device functions****
__global__ void filter_kernel(float *image, float *denoised_image, float *gauss_kernel, int p_size, int im_size, float f_s );
__device__ void find_patch(float *pixel_patch, int current_pixel,int im_size,int p_size,float *image);
__device__ float compute_weight(float *pixel_patch,float *temp_patch,float *g_kernel, int p_size, float f_sigma);


int main(int argc, char *argv[]){

  //pass all parameters needed for computation.
  //patch size now is defined because must be known in compile time
  if(argc !=5){
    printf("Give correct arguments:1)source file(.csv format), 2) image size, 3)filter sigma, 4)gaussian filter sigma\n");
    exit(1);
  }

  char *image_path      = argv[1];
  int   image_size      = atoi(argv[2]);
  float f_sigma         = atof(argv[3]);
  float g_sigma         = atof(argv[4]);
  int   patch_size      = PSIZE;
  float *den_image      = (float *)malloc(image_size*image_size*sizeof(float));

  //read 2-D image
  float **I = read_image(image_path,image_size);

  //add padding
  int padded_size;
  float **D=add_padding(I,image_size,patch_size,&padded_size);

  //the initial 2-D image is now useless
  for(int i=0;i<image_size;i++)free(I[i]);
  free(I);

  //write image to rowmajor format
  float *padded_I = to_rowmajor(D,padded_size);

  //the padded 2-D image is now useless
  for(int i=0;i<padded_size;i++)free(D[i]);
  free(D);

  //compute gaussian filte
  float *GKERNEL = gaussian_filter(patch_size,g_sigma);

  //******** gpu section ********

  //define block and grid size
  int block_size = 256;
  int grid_size = image_size*image_size/block_size;

  //device variables
  float *dev_padded_image, *dev_denoised_image, *dev_g_kernel;

  //variables to compute time of execution.
  hipEvent_t start,stop;

  //memory allocation in device memory

  checkCuda(hipEventCreate(&start));
  checkCuda(hipEventCreate(&stop));

  checkCuda(hipMalloc((void **)&dev_g_kernel,patch_size*patch_size*sizeof(float)));
  checkCuda(hipMemcpy(dev_g_kernel,GKERNEL,patch_size*patch_size*sizeof(float),hipMemcpyHostToDevice));

  checkCuda(hipMalloc((void **)&dev_padded_image,padded_size*padded_size*sizeof(float)));
  checkCuda(hipMemcpy(dev_padded_image,padded_I,padded_size*padded_size*sizeof(float),hipMemcpyHostToDevice));

  checkCuda(hipMalloc((void **)&dev_denoised_image,image_size*image_size*sizeof(float)));

  /*************************nlm execution*************************/
  hipEventRecord(start);
  filter_kernel<<<grid_size,block_size>>>(dev_padded_image,dev_denoised_image,dev_g_kernel,patch_size,padded_size,f_sigma);
  hipEventRecord(stop);
  /***************************************************************/

  checkCuda(hipMemcpy(den_image,dev_denoised_image,image_size*image_size*sizeof(float),hipMemcpyDeviceToHost));

  checkCuda(hipEventSynchronize(stop));
  float milliseconds = 0;
  checkCuda(hipEventElapsedTime(&milliseconds,start,stop));


  checkCuda(hipEventDestroy(start));
  checkCuda(hipEventDestroy(stop));
  checkCuda(hipFree(dev_g_kernel));
  checkCuda(hipFree(dev_padded_image));
  checkCuda(hipFree(dev_denoised_image));

  //some print messages
  printf("*************************************************************************************\n");
  printf("gpu execution (without use of shared memory) finished total time is %f msec\n",milliseconds);
  printf("image size is %d\n", image_size);
  printf("patch size is %d\n", patch_size);
  printf("filtes sigma is %f and gauss kernel sigma is %f\n",f_sigma,g_sigma);
  printf("**************************************************************************************\n");

  char denoised_path[200];
  snprintf(denoised_path,sizeof(denoised_path),"../data/gpu_denoised_image_%d_%d.csv",image_size,patch_size);
  write_image2(denoised_path,den_image,image_size);

  free(GKERNEL);
  free(den_image);
  free(padded_I);
}



/***********host functions****************/

float **add_padding(float **image , int im_size, int p_size, int *pad_size){

  int padding     = (p_size-1)/2;
  int padded_size = im_size + padding*2;
  *pad_size = padded_size;

  //allocate memomry for padded image
  float **padded_image =(float **)malloc(padded_size*sizeof(float *));
  for(int i=0;i<padded_size;i++)padded_image[i]=(float *)malloc(padded_size*sizeof(float));

  //initialize all values to -1 for debugging puproses...Later this will not be necessary
  for(int i=0;i<padded_size;i++){
    for(int j=0;j<padded_size;j++){
      padded_image[i][j]=-1;
    }
  }

  //first copy the body of the padded array which will remain the same
  //and also compute outter pixels through mirroring.
  for(int row = padding;row<padding+im_size;row++){
    for(int col=0;col<padded_size;col++){
      if(col<padding)padded_image[row][col] = image[row-padding][padding-col-1];
      else if(col>=padding+im_size)padded_image[row][col] = image[row-padding][im_size-1-(col-padding-im_size)];
      else padded_image[row][col] = image[row-padding][col-padding];
    }
  }

  //mirroring first and last rows.
  for(int row=0;row<padding;row++){
    for(int col = 0;col<padded_size;col++)padded_image[row][col] = padded_image[padding+(padding-row-1)][col];
  }
  for(int row=padding+im_size;row<padded_size;row++){
    for(int col=0;col<padded_size;col++)padded_image[row][col] = padded_image[padding+im_size-1-(row-padding-im_size)][col];
  }


  return padded_image;
}


float *gaussian_filter(int patch_size, float sigma){

  //allocate memory for the kernel
  float *kernel   = (float *)malloc(patch_size*patch_size*sizeof(float ));

  //compute variance
  float s         = 2.0*sigma*sigma;

  //sum for normalization
  float sum       = 0.0;

  int boundary    = patch_size/2;
  float r;

  for(int x=-boundary; x<=boundary; x++){
    for(int y=-boundary; y<=boundary; y++){
      r = x*x + y*y;
      kernel[(x+boundary)*patch_size+(y+boundary)] = exp(-r/s);
      sum += kernel[(x+boundary)*patch_size+(y+boundary)];
    }
  }

  //find the max and alos divide with the total sum
  float max = -1;
  for(int x=0;x<patch_size;x++){
    for(int y=0;y<patch_size;y++){
      kernel[x*patch_size+y]= kernel[x*patch_size+y]/sum;
      if(kernel[x*patch_size+y]>max) max = kernel[x*patch_size+y];
    }
  }

  //devide also with the max value
  for(int x=0;x<patch_size;x++){
    for(int y=0;y<patch_size;y++){
      kernel[x*patch_size+y]= kernel[x*patch_size+y]/max;
    }
  }


  return kernel;
}


float *to_rowmajor(float **image, int im_size){

  //allocate memory
  float *row_image = (float *)malloc(im_size*im_size*sizeof(float));

  //copy pixels to 1-D array.
  for(int row=0;row<im_size;row++){
    for(int col=0;col<im_size;col++){
      row_image[row*im_size+col] = image[row][col];
    }
  }

  return row_image;
}


void checkCuda(hipError_t result){
  if(result != hipSuccess){
    printf("CUDA Runtime Error :%s\n",hipGetErrorString(result));
    exit(1);
  }
}


float **read_image(char *im_path,int im_size){

  //open image file
  FILE *file = fopen(im_path,"r");

  //allocate memory
  float **image = (float **)malloc(im_size*sizeof(float *));
  for(int i=0;i<im_size;i++)image[i] = (float *)malloc(im_size*sizeof(float));

  //read the image
  for(int i=0;i<im_size;i++){
    for(int j=0;j<im_size;j++){
      fscanf(file,"%f,",image[i]+j);
    }
  }

  fclose(file);
  return image;
}


void   write_image2(char *d_path , float  *im, int im_size){

  //open file
  FILE *file = fopen(d_path,"w");

  //write image to file
  for(int i=0;i<im_size;i++){
    for(int j=0;j<im_size;j++){
      fprintf(file,"%f,",im[i*im_size +j]);
    }
    fprintf(file,"\n");
  }

  fclose(file);
}


/***********device functions****************/

__global__ void filter_kernel(float *image, float *denoised_image, float *gauss_kernel, int p_size, int im_size, float f_s ){

  int id = blockIdx.x*blockDim.x+threadIdx.x;

  //compute the pixel which corresponds to each thread
  int padded_pixel  = im_size*(p_size-1)/2+blockDim.x/(im_size-2*(p_size-1)/2)*blockIdx.x*im_size+
                      im_size*(threadIdx.x/(im_size-2*(p_size-1)/2))+
                      threadIdx.x%(im_size-2*(p_size-1)/2)+(p_size-1)/2;


  int boundary1 = (p_size-1)/2;
  int boundary2 = im_size - (p_size-1)/2;

  float pixel_patch[PSIZE*PSIZE];
  float temp_patch[PSIZE*PSIZE];

  int temp_pixel;
  float W;
  float Z   = 0;
  float estimation = 0;

  //each thread computes its own patch
  find_patch(pixel_patch,padded_pixel,im_size,p_size,image);

  for(int row=boundary1;row<boundary2;row++){
    for(int col=boundary1;col<boundary2;col++){
      temp_pixel = row *im_size + col;
      find_patch(temp_patch,temp_pixel,im_size,p_size,image);
      W = compute_weight(pixel_patch,temp_patch,gauss_kernel,p_size,f_s);
      Z = Z + W;
      estimation = estimation + W*image[temp_pixel];
    }
  }
  denoised_image[id] = estimation/Z;
}


__device__ void find_patch(float *pixel_patch, int current_pixel,int im_size,int p_size,float *image){
  int boundary = (p_size-1)/2;
  for(int i=-boundary;i<=boundary;i++){
    for(int j=-boundary;j<=boundary;j++){
      pixel_patch[(i+boundary)*p_size+(j+boundary)] = image[current_pixel+i*im_size+j];
    }
  }
}


__device__ float compute_weight(float *pixel_patch,float *temp_patch,float *g_kernel, int p_size, float f_sigma){
  float temp=0;
  for(int i=0;i<p_size*p_size;i++) temp = temp + g_kernel[i]*g_kernel[i]*(pixel_patch[i]-temp_patch[i])*(pixel_patch[i]-temp_patch[i]);
  return exp(-temp/(f_sigma*f_sigma));
}
